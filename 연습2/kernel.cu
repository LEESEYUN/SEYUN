#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hip/device_functions.h>

#include <stdio.h>

#define MAX 33*1024
const int threadPerblock = 256;
const int gridPerblock = (MAX + threadPerblock - 1) / threadPerblock;



__global__ void multi(float*a, float*b, float*c){
	__shared__ float cache[threadPerblock];
	
	int tid = threadIdx.x + blockIdx.x*blockDim.x;
	int cacheIndex = threadIdx.x;
	float temp = 0;
	while (tid < MAX){
		temp = b[tid] * a[tid];
		tid += gridDim.x*blockDim.x;
	}
	cache[cacheIndex] = temp;
 

	__syncthreads();

	int i = blockDim.x / 2;
	while (i != 0){
		if (cacheIndex < i)
			cache[cacheIndex] += cache[cacheIndex + i];
		__syncthreads();
		i /= 2;
	}
	if (cacheIndex == 0)
		c[blockIdx.x] = cache[0];

}



int main()
{
	float a[MAX],b[MAX],c[gridPerblock];
	float* dev_a, *dev_b, *dev_c;


	for (int i = 0; i < MAX; i++){
		a[i] = i;
		b[i] = -i;
	}


	hipMalloc((void**)&dev_a, sizeof(float)*MAX);
	hipMalloc((void**)&dev_b, sizeof(float)*MAX);
	hipMalloc((void**)&dev_c, sizeof(float)*gridPerblock);

	hipMemcpy(dev_a, a, sizeof(float)*MAX, hipMemcpyHostToDevice);
	hipMemcpy(dev_b, b, sizeof(float)*MAX, hipMemcpyHostToDevice);

	multi << <threadPerblock, threadPerblock >> >(dev_a, dev_b, dev_c);

	hipMemcpy(c, dev_c, sizeof(float)*gridPerblock, hipMemcpyDeviceToHost);

	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);
	
	float temp=0;
	for (int i = 0; i < gridPerblock; i++){
		//temp += c[i];
		printf("%f\n", c[i]);
		/*
		if (i % 10 == 0){
			printf("\n");
		}
		*/
	}
	//printf("%f", temp);

	
    return 0;
}

