#include "hip/hip_runtime.h"
#include <iostream>
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#ifdef __INTELLISENSE__
void __syncthreads();
#endif


#define ThreadPerBlock 1024

__global__ void dot(float*a, float*b, float*c, int threadperblock, int max){
	__shared__ float cache[ThreadPerBlock];
	int tid = threadIdx.x + blockDim.x*blockIdx.x;
	float temp = 0;
	int cacheindex = threadIdx.x;
	while (tid < max){
		temp = a[tid] * b[tid];
		tid += gridDim.x*blockDim.x;
	}
	cache[cacheindex] = temp;

	__syncthreads();
	int i = blockDim.x / 2;
	while (i != 0){
		if (cacheindex < i)
			cache[cacheindex] += cache[cacheindex + i];
		__syncthreads();
		i /= 2;
	}
	if (cacheindex == 0)
		c[blockIdx.x] = cache[0];



}




extern "C" void kernel(float *a, float *b, float *c, int max, int threadperblock){
	const int BlockPerGrid = (max + threadperblock - 1) / threadperblock;
	float *dev_a, *dev_b, *dev_c;
	for (int i = 0; i < max; i++){
		a[i] = (float)i;
		b[i] = (float)i*i;
	}

	hipMalloc((void**)&dev_a, sizeof(float)*max);
	hipMalloc((void**)&dev_b, sizeof(float)*max);
	hipMalloc((void**)&dev_c, sizeof(float)*BlockPerGrid);

	hipMemcpy(dev_a, a, sizeof(float)*max, hipMemcpyHostToDevice);
	hipMemcpy(dev_b, b, sizeof(float)*max, hipMemcpyHostToDevice);

	dot<< <BlockPerGrid, threadperblock >> >(dev_a, dev_b, dev_c,threadperblock,max);

	hipMemcpy(c, dev_c, sizeof(float)*BlockPerGrid, hipMemcpyDeviceToHost);
	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);

}


